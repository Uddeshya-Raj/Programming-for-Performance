
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h> 
#include <string.h>
#include <time.h>
#include <vector>
#include <algorithm>

using namespace std;

#define NSEC_SEC_MUL (1.0e9) 

#define cudaCheckError(ans){ gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

struct timespec begin_grid, end_main;

int pnts = 0;

__device__ double global_storage[12000][10];
__device__ int counter;

__device__ void recursive_func(double *grid, double *c, double *e, double *x, double *sums, int index){
  if(index >= 10){
    bool good_sum = true;
    for(int k=0;k<10;k++){
      if(fabs(sums[k]) > e[k]){
        good_sum = false;
        break;
      }
    }
    if(good_sum){
      int idx = atomicAdd(&counter,1);
      global_storage[idx][0] = x[0];
      global_storage[idx][1] = x[1];
      global_storage[idx][2] = x[2];
      global_storage[idx][3] = x[3];
      global_storage[idx][4] = x[4];
      global_storage[idx][5] = x[5];
      global_storage[idx][6] = x[6];
      global_storage[idx][7] = x[7];
      global_storage[idx][8] = x[8];
      global_storage[idx][9] = x[9];
    }
    return;
  }

  double start = grid[index*3+0];
  double end = grid[index*3+1];
  double step = grid[index*3+2];
  for(double xi=start; xi<end; xi+=step){
    double temp[10] = {0.0f};
    bool bad_i = false;
    for(int j=0;j<10;j++){
      temp[j] = sums[j]+c[j*10+index]*xi;
      if(temp[j]>e[j]){
        bad_i = true;
        break;
      }
    }
    if(bad_i) continue;
    else{
      double new_x[10];
      for(int i=0;i<10;i++) new_x[i]=x[i];
      new_x[index] = xi;
      recursive_func(grid, c, e, new_x, temp, index+1);
    }
  }
}

__global__ void wrapper_kernel(double *grid, double *c, double *e, double *x, double *sums, int index){

  if(blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0){
    
  }

  float x0 = grid[0*3+2]*blockIdx.x + grid[0*3+0]; //using x direction for mimicing 0th level of loop
  float x1 = grid[1*3+2]*blockIdx.y + grid[1*3+0]; //using y direction for mimicing 1st level of loop
  float x2 = grid[2*3+2]*blockIdx.z + grid[2*3+0]; //using z direction for mimicing 2nd level of loop
  // check if current 3 values of x are effective or not
  double temp[10] = {0.0f};
  bool bad_values = false;
  for(int i=0;i<10;i++){
    temp[i] = sums[i] + c[i*10+0]*x0 + c[i*10+1]*x1 + c[i*10+2]*x2;
    if(temp[i] > e[i]){
      bad_values = true;
      break;
    }
  }
  if(!bad_values){
    // int idx = atomicAdd(&counter, 1);
    double new_x[10] = {0.0f};
    for(int i=0;i<10;i++) new_x[i] = x[i];
    new_x[0] = x0;
    new_x[1] = x1;
    new_x[2] = x2;
    recursive_func(grid, c, e, new_x, temp, index+3);
  }
}

// original call : func(grid, c, e, x[0,0,0,...], sums[-d0,-d1,-d2,...], 0)
void func(double (*grid)[3], double (*c)[10], double *e, double *x, double *sums, int index, FILE* fptr){
  if(index >= 10){
    bool good_sum = true;
    for(int k=0;k<10;k++){
      if(fabs(sums[k]) > e[k]){
        good_sum = false;
        break;
      }
    }
    if(good_sum){
      pnts += 1;
      fprintf(fptr, "%lf\t", x[0]);
      fprintf(fptr, "%lf\t", x[1]);
      fprintf(fptr, "%lf\t", x[2]);
      fprintf(fptr, "%lf\t", x[3]);
      fprintf(fptr, "%lf\t", x[4]);
      fprintf(fptr, "%lf\t", x[5]);
      fprintf(fptr, "%lf\t", x[6]);
      fprintf(fptr, "%lf\t", x[7]);
      fprintf(fptr, "%lf\t", x[8]);
      fprintf(fptr, "%lf\n", x[9]);
    }
    return;
  }  
  double start = grid[index][0];
  double end = grid[index][1];
  double step = grid[index][2];
  for(double xi=start;xi<end;xi+=step){
    double temp[10] = {0.0f};
    bool bad_i = false;
    for(int j=0;j<10;j++){
      temp[j] = sums[j]+c[j][index]*xi;
      if(temp[j]>e[j]){
        bad_i = true;
        break;
      }
    }
    if(bad_i) continue;
    else{
      double new_x[10] = {0.0f};
      for(int i=0;i<10;i++) new_x[i] = x[i];
      new_x[index] = xi;
      func(grid, c, e, new_x, temp, index+1, fptr);
    }
  }
}

// to store values of disp.txt
double a[120];

// to store values of grid.txt
double b[30];

int main() {

  hipError_t err;
  size_t newStackSize = 4096; // <--- adjusted stack size
  err = hipDeviceSetLimit(hipLimitStackSize, newStackSize);
  if (err != hipSuccess) {
      std::cerr << "Error setting stack size: " << hipGetErrorString(err) << std::endl;
      return -1;
  }

  size_t currentStackSize;
  err = hipDeviceGetLimit(&currentStackSize, hipLimitStackSize); 
  if (err != hipSuccess) {
      std::cerr << "Error getting stack size limit: " << hipGetErrorString(err) << std::endl;
      return -1;
  }
  std::cout << "New stack size limit: " << currentStackSize << " bytes" << std::endl;

  int i, j;

  i = 0;
  FILE* fp = fopen("./disp.txt", "r");
  if (fp == NULL) {
    printf("Error: could not open file\n");
    return 1;
  }

  while (!feof(fp)) {
    if (!fscanf(fp, "%lf", &a[i])) {
      printf("Error: fscanf failed while reading disp.txt\n");
      exit(EXIT_FAILURE);
    }
    i++;
  }
  fclose(fp);

  // read grid file
  j = 0;
  FILE* fpq = fopen("./grid.txt", "r");
  if (fpq == NULL) {
    printf("Error: could not open file\n");
    return 1;
  }

  while (!feof(fpq)) {
    if (!fscanf(fpq, "%lf", &b[j])) {
      printf("Error: fscanf failed while reading grid.txt\n");
      exit(EXIT_FAILURE);
    }
    j++;
  }
  fclose(fpq);

  hipMemset(&counter, 0, sizeof(int));
  
  double kk = 0.3;

  int pos = 0;
  double grid[10][3], *d_grid;
  for(int i=0;i<10;i++){
    for(int j=0;j<3;j++){
      grid[i][j] = b[pos++];
    }
  }

  pos = 0;
  double c[10][10], *d_c;
  double d[10];
  double ey[10];
  double e[10], d_e[10];

  for(int i = 0; i<10; i++){
    for(int j=0; j<10; j++){
      c[i][j] = a[pos++];
    }
    d[i] = a[pos++];
    ey[i] = a[pos++];
    e[i] = ey[i]*kk;
  }

  double x[10] = {0.0f}, d_x[10];
  double sums[10], d_sums[10];
  for(int i=0;i<10;i++) sums[i] = -d[i];

  cudaCheckError(hipMalloc((void**)&d_grid, 30*sizeof(double)));
  cudaCheckError(hipMemcpy(d_grid, grid, sizeof(double)*30, hipMemcpyHostToDevice));

  cudaCheckError(hipMalloc((void**)&d_c, 100*sizeof(double)));
  cudaCheckError(hipMemcpy(d_c, c, sizeof(double)*100, hipMemcpyHostToDevice));

  cudaCheckError(hipMalloc((void**)&d_e, sizeof(double)*10));
  cudaCheckError(hipMemcpy(d_e, e, sizeof(double)*10, hipMemcpyHostToDevice));

  cudaCheckError(hipMalloc((void**)&d_x, sizeof(double)*10));
  cudaCheckError(hipMemcpy(d_x, x, sizeof(double)*10, hipMemcpyHostToDevice));

  cudaCheckError(hipMalloc((void**)&d_sums, sizeof(double)*10));
  cudaCheckError(hipMemcpy(d_sums, sums, sizeof(double)*10, hipMemcpyHostToDevice));


  FILE *fptr = fopen("./results-v1.txt", "w");
  if(fptr == NULL){
    printf("Error in creating file !");
    exit(1);
  }
  // clock_gettime(CLOCK_MONOTONIC_RAW, &begin_grid);
  // func(grid, c, e, x, sums, 0, fptr);  
  // clock_gettime(CLOCK_MONOTONIC_RAW, &end_main);

  dim3 blockpergird((grid[0][1]-grid[0][0])/grid[0][2], (grid[1][1]-grid[1][0])/grid[1][2], (grid[2][1]-grid[2][0])/grid[2][2]);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  // wrapper_kernel(double (*grid)[3], double (*c)[10], double *e, double *x, double *sums, int index)
  hipEventRecord(start,0);
  wrapper_kernel<<<blockpergird, 1>>>(d_grid, d_c, d_e, d_x, d_sums, 0);
  hipEventRecord(end,0);
  hipEventSynchronize(end);
  float kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time,start,end);
  // cudaCheckError(cudaPeekAtLastError());
  // cudaCheckError(cudaDeviceSynchronize());

  int h_counter = 0;
  double (*output)[10];
  output = (double(*)[10])malloc(12000 * 10 * sizeof(double));
  cudaCheckError(hipMemcpyFromSymbol(&h_counter, HIP_SYMBOL(counter), sizeof(int), 0, hipMemcpyDeviceToHost));
  cudaCheckError(hipMemcpyFromSymbol(output, HIP_SYMBOL(global_storage), sizeof(double)*h_counter*10, 0, hipMemcpyDeviceToHost));

  vector<vector<double>> data(h_counter, vector<double>(10));
  for (int i = 0; i < h_counter; i++) {
      for (int j = 0; j < 10; j++) {
          data[i][j] = output[i][j];
      }
  }

  std::sort(data.begin(), data.end());

  for (const auto& row : data) {
      for (const auto& value : row) {
          fprintf(fptr, "%lf\t", value);
      }
      fprintf(fptr, "\n");
  }
  fclose(fptr);

  // printf("result pnts (CPU) : %d \n", pnts);
  printf("result pnts (GPU) : %d \n", h_counter);
  // printf("cpu time = %f seconds\n",
  //        (end_main.tv_nsec - begin_grid.tv_nsec) / NSEC_SEC_MUL +
  //            (end_main.tv_sec - begin_grid.tv_sec));
  printf("kernel time = %f seconds\n",kernel_time/1000);
  return EXIT_SUCCESS;
}

